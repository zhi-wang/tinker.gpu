#include "hip/hip_runtime.h"
#include "add.h"
#include "launch.h"
#include "lpiston.h"
#include "mdcalc.h"
#include "mdegv.h"
#include "mdpq.h"
#include "nose.h"
#include "rattle.h"
#include <tinker/detail/units.hh>


namespace tinker {
template <class HTYPE>
__global__
void constrain_methyl_cu1(
   double eps, int nratch2, const int (*restrict iratch2)[3],
   const pos_prec (*restrict kratch2)[2], int nratch3,
   const int (*restrict iratch3)[4], const pos_prec (*restrict kratch3)[3],

   time_prec dt, pos_prec* restrict xnew, pos_prec* restrict ynew,
   pos_prec* restrict znew, const pos_prec* restrict xold,
   const pos_prec* restrict yold, const pos_prec* restrict zold,
   const double* restrict massinv,

   double rats1,

   vel_prec* restrict vx, vel_prec* restrict vy, vel_prec* restrict vz)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;
   constexpr int maxiter = 500;


   int n23 = nratch2 + nratch3;
   for (int im0 = ithread; im0 < n23; im0 += stride) {
      bool methyl = im0 >= nratch2;
      int ia, ib, ic, id;
      double lab, lac, lad;
      double rma, rmb, rmc, rmd;
      if (methyl) {
         int im = im0 - nratch2;
         ia = iratch3[im][0];
         ib = iratch3[im][1];
         ic = iratch3[im][2];
         id = iratch3[im][3];
         lab = kratch3[im][0];
         lac = kratch3[im][1];
         lad = kratch3[im][2];
         rmd = massinv[id];
      } else {
         int im = im0;
         ia = iratch2[im][0];
         ib = iratch2[im][1];
         ic = iratch2[im][2];
         lab = kratch2[im][0];
         lac = kratch2[im][1];
      }
      rma = massinv[ia];
      rmb = massinv[ib];
      rmc = massinv[ic];
      double ae1 = 1.0, be1 = 1.0, ce1 = 1.0, de1 = 1.0;
      if CONSTEXPR (eq<HTYPE, LPRAT>()) {
         ae1 = rats1;
         be1 = rats1;
         ce1 = rats1;
         rma /= ae1;
         rmb /= be1;
         rmc /= ce1;
         if (methyl) {
            de1 = rats1;
            rmd /= de1;
         }
      }


      // vectors AB0, AB1;
      double xb0, yb0, zb0, xb1, yb1, zb1;
      xb0 = xold[ib] - xold[ia];
      yb0 = yold[ib] - yold[ia];
      zb0 = zold[ib] - zold[ia];
      xb1 = xnew[ib] - xnew[ia];
      yb1 = ynew[ib] - ynew[ia];
      zb1 = znew[ib] - znew[ia];


      // vectors AC0, AC1;
      double xc0, yc0, zc0, xc1, yc1, zc1;
      xc0 = xold[ic] - xold[ia];
      yc0 = yold[ic] - yold[ia];
      zc0 = zold[ic] - zold[ia];
      xc1 = xnew[ic] - xnew[ia];
      yc1 = ynew[ic] - ynew[ia];
      zc1 = znew[ic] - znew[ia];


      // vectors AD0, AD1
      double xd0, yd0, zd0, xd1, yd1, zd1;
      if (methyl) {
         xd0 = xold[id] - xold[ia];
         yd0 = yold[id] - yold[ia];
         zd0 = zold[id] - zold[ia];
         xd1 = xnew[id] - xnew[ia];
         yd1 = ynew[id] - ynew[ia];
         zd1 = znew[id] - znew[ia];
      }


      double dxa = 0, dya = 0, dza = 0;
      double dxb = 0, dyb = 0, dzb = 0;
      double dxc = 0, dyc = 0, dzc = 0;
      double dxd = 0, dyd = 0, dzd = 0;


      int iter = 0;
      bool done = false;
      while (not done and iter < maxiter) {
         ++iter;
         done = true;
         double x1, y1, z1, dot, delta, term;


         // AB
         x1 = xb1 + dxb - dxa;
         y1 = yb1 + dyb - dya;
         z1 = zb1 + dzb - dza;
         delta = x1 * x1 + y1 * y1 + z1 * z1 - lab * lab;
         if (fabs(delta) > eps) {
            dot = xb0 * x1 + yb0 * y1 + zb0 * z1;
            term = 0.5 * delta / ((rma + rmb) * dot);
            dxa += term * xb0 * rma;
            dya += term * yb0 * rma;
            dza += term * zb0 * rma;
            dxb -= term * xb0 * rmb;
            dyb -= term * yb0 * rmb;
            dzb -= term * zb0 * rmb;
            done = false;
         }


         // AC
         x1 = xc1 + dxc - dxa;
         y1 = yc1 + dyc - dya;
         z1 = zc1 + dzc - dza;
         delta = x1 * x1 + y1 * y1 + z1 * z1 - lac * lac;
         if (fabs(delta) > eps) {
            dot = xc0 * x1 + yc0 * y1 + zc0 * z1;
            term = 0.5 * delta / ((rma + rmc) * dot);
            dxa += term * xc0 * rma;
            dya += term * yc0 * rma;
            dza += term * zc0 * rma;
            dxc -= term * xc0 * rmc;
            dyc -= term * yc0 * rmc;
            dzc -= term * zc0 * rmc;
            done = false;
         }


         // AD
         if (methyl) {
            x1 = xd1 + dxd - dxa;
            y1 = yd1 + dyd - dya;
            z1 = zd1 + dzd - dza;
            delta = x1 * x1 + y1 * y1 + z1 * z1 - lad * lad;
            if (fabs(delta) > eps) {
               dot = xd0 * x1 + yd0 * y1 + zd0 * z1;
               term = 0.5 * delta / ((rma + rmd) * dot);
               dxa += term * xd0 * rma;
               dya += term * yd0 * rma;
               dza += term * zd0 * rma;
               dxd -= term * xd0 * rmd;
               dyd -= term * yd0 * rmd;
               dzd -= term * zd0 * rmd;
               done = false;
            }
         }
      }


      xnew[ia] += dxa;
      ynew[ia] += dya;
      znew[ia] += dza;
      xnew[ib] += dxb;
      ynew[ib] += dyb;
      znew[ib] += dzb;
      xnew[ic] += dxc;
      ynew[ic] += dyc;
      znew[ic] += dzc;
      if (methyl) {
         xnew[id] += dxd;
         ynew[id] += dyd;
         znew[id] += dzd;
      }
      if CONSTEXPR (not eq<HTYPE, SHAKE>()) {
         double invdt = 1 / dt;
         vx[ia] += dxa * invdt * ae1;
         vy[ia] += dya * invdt * ae1;
         vz[ia] += dza * invdt * ae1;
         vx[ib] += dxb * invdt * be1;
         vy[ib] += dyb * invdt * be1;
         vz[ib] += dzb * invdt * be1;
         vx[ic] += dxc * invdt * ce1;
         vy[ic] += dyc * invdt * ce1;
         vz[ic] += dzc * invdt * ce1;
         if (methyl) {
            vx[id] += dxd * invdt * de1;
            vy[id] += dyd * invdt * de1;
            vz[id] += dzd * invdt * de1;
         }
      }
   }
}


void rattle_methyl_cu(time_prec dt, const pos_prec* xold, const pos_prec* yold,
                      const pos_prec* zold)
{
   int n23 = nratch2 + nratch3;
   if (n23 <= 0)
      return;


   auto ker = constrain_methyl_cu1<RATTLE>;
   launch_k2s(g::s0, 64, n23, ker,

              rateps, nratch2, iratch2, kratch2, nratch3, iratch3, kratch3,

              dt, xpos, ypos, zpos, xold, yold, zold, massinv,

              lp_rats1,

              vx, vy, vz);
}


void lprat_methyl_cu(time_prec dt, const pos_prec* xold, const pos_prec* yold,
                     const pos_prec* zold)
{
   int n23 = nratch2 + nratch3;
   if (n23 <= 0)
      return;


   auto ker = constrain_methyl_cu1<LPRAT>;
   launch_k2s(g::s0, 64, n23, ker,

              rateps, nratch2, iratch2, kratch2, nratch3, iratch3, kratch3,

              dt, xpos, ypos, zpos, xold, yold, zold, massinv,

              lp_rats1,

              vx, vy, vz);
}


void shake_methyl_cu(time_prec dt, pos_prec* xnew, pos_prec* ynew,
                     pos_prec* znew, const pos_prec* xold, const pos_prec* yold,
                     const pos_prec* zold)
{
   int n23 = nratch2 + nratch3;
   if (n23 <= 0)
      return;


   auto ker = constrain_methyl_cu1<SHAKE>;
   launch_k2s(g::s0, 64, n23, ker,

              rateps, nratch2, iratch2, kratch2, nratch3, iratch3, kratch3,

              dt, xnew, ynew, znew, xold, yold, zold, massinv,

              lp_rats1,

              nullptr, nullptr, nullptr);
}


template <bool DO_V>
__global__
void constrain2_methyl_cu1(int nratch2, const int (*restrict iratch2)[3],
                           int nratch3, const int (*restrict iratch3)[4],

                           time_prec dt, vel_prec* restrict vx,
                           vel_prec* restrict vy, vel_prec* restrict vz,
                           virial_buffer restrict vir_buf,

                           const pos_prec* restrict xpos,
                           const pos_prec* restrict ypos,
                           const pos_prec* restrict zpos,
                           const double* restrict massinv)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;


   const double vterm = 2 / (dt * units::ekcal);
   double vxx, vyx, vzx, vyy, vzy, vzz;
   if CONSTEXPR (DO_V) {
      vxx = 0, vyx = 0, vzx = 0, vyy = 0, vzy = 0, vzz = 0;
   }


   int n23 = nratch2 + nratch3;
   for (int im0 = ithread; im0 < n23; im0 += stride) {
      bool methyl = im0 >= nratch2;
      int ia, ib, ic, id;
      double rma, rmb, rmc, rmd;
      if (methyl) {
         int im = im0 - nratch2;
         ia = iratch3[im][0];
         ib = iratch3[im][1];
         ic = iratch3[im][2];
         id = iratch3[im][3];
         rmd = massinv[id];
      } else {
         int im = im0;
         ia = iratch2[im][0];
         ib = iratch2[im][1];
         ic = iratch2[im][2];
      }
      rma = massinv[ia];
      rmb = massinv[ib];
      rmc = massinv[ic];


      // matrix form
      // (mab AB3.AB3   rma AB3.AC3  rma AB3.AD3) (lb) = (AB3.vAB0)
      // (rma AC3.AB3   mac AC3.AC3  rma AC3.AD3) (lc) = (AC3.vAC0)
      // (rma AD3.AB3   rma AD3.AC3  mad AD3.AD3) (ld) = (AD3.vAD0)


      // vectors AB3, vAB0, AB3 dot vAB0
      double xb3, yb3, zb3, vxb, vyb, vzb, dotb, rb2;
      xb3 = xpos[ib] - xpos[ia];
      yb3 = ypos[ib] - ypos[ia];
      zb3 = zpos[ib] - zpos[ia];
      vxb = vx[ib] - vx[ia];
      vyb = vy[ib] - vy[ia];
      vzb = vz[ib] - vz[ia];
      dotb = xb3 * vxb + yb3 * vyb + zb3 * vzb;
      rb2 = xb3 * xb3 + yb3 * yb3 + zb3 * zb3;
      // vectors AC3, vAC0, AC3 dot vAC0
      double xc3, yc3, zc3, vxc, vyc, vzc, dotc, rc2;
      xc3 = xpos[ic] - xpos[ia];
      yc3 = ypos[ic] - ypos[ia];
      zc3 = zpos[ic] - zpos[ia];
      vxc = vx[ic] - vx[ia];
      vyc = vy[ic] - vy[ia];
      vzc = vz[ic] - vz[ia];
      dotc = xc3 * vxc + yc3 * vyc + zc3 * vzc;
      rc2 = xc3 * xc3 + yc3 * yc3 + zc3 * zc3;
      // AB3 dot AC3
      double dotbc = xb3 * xc3 + yb3 * yc3 + zb3 * zc3;
      // vectors AD3, vAD0, AD3 dot vAD0
      // AB3 dot AD3, AC3 dot AD3
      double xd3 = 0, yd3 = 0, zd3 = 0, vxd, vyd, vzd, dotd, rd2;
      double dotbd, dotcd;
      if (methyl) {
         xd3 = xpos[id] - xpos[ia];
         yd3 = ypos[id] - ypos[ia];
         zd3 = zpos[id] - zpos[ia];
         vxd = vx[id] - vx[ia];
         vyd = vy[id] - vy[ia];
         vzd = vz[id] - vz[ia];
         dotd = xd3 * vxd + yd3 * vyd + zd3 * vzd;
         rd2 = xd3 * xd3 + yd3 * yd3 + zd3 * zd3;
         dotbd = xb3 * xd3 + yb3 * yd3 + zb3 * zd3;
         dotcd = xc3 * xd3 + yc3 * yd3 + zc3 * zd3;
      }


      double lb, lc, ld;
      double m11, m12, m22; // m21 = m12
      double m13, m23, m33; // m31 = m13, m32 = m23
      double det;
      m11 = (rma + rmb) * rb2;
      m12 = rma * dotbc;
      m22 = (rma + rmc) * rc2;
      if (not methyl) {
         det = m11 * m22 - m12 * m12;
         det = 1 / det;
         lb = (m22 * dotb - m12 * dotc) * det;
         lc = (m11 * dotc - m12 * dotb) * det;
         ld = 0;
      } else {
         m13 = rma * dotbd;
         m23 = rma * dotcd;
         m33 = (rma + rmd) * rd2;
         det = (m11 * m22 - m12 * m12) * m33 + (m12 * m13 - m11 * m23) * m23 +
            (m12 * m23 - m22 * m13) * m13;
         det = 1 / det;
         double i11 = m22 * m33 - m23 * m23;
         double i22 = m11 * m33 - m13 * m13;
         double i33 = m11 * m22 - m12 * m12;
         double i12 = m13 * m23 - m12 * m33;
         double i13 = m12 * m23 - m13 * m22;
         double i23 = m12 * m13 - m11 * m23;
         lb = (i11 * dotb + i12 * dotc + i13 * dotd) * det;
         lc = (i12 * dotb + i22 * dotc + i23 * dotd) * det;
         ld = (i13 * dotb + i23 * dotc + i33 * dotd) * det;
      }


      lb = -lb;
      lc = -lc;
      ld = -ld;
      double xtermb, ytermb, ztermb, xtermc, ytermc, ztermc;
      xtermb = xb3 * lb;
      ytermb = yb3 * lb;
      ztermb = zb3 * lb;
      xtermc = xc3 * lc;
      ytermc = yc3 * lc;
      ztermc = zc3 * lc;
      double xtermd = 0, ytermd = 0, ztermd = 0;
      if (methyl) {
         xtermd = xd3 * ld;
         ytermd = yd3 * ld;
         ztermd = zd3 * ld;
      }
      vx[ia] -= (xtermb + xtermc + xtermd) * rma;
      vy[ia] -= (ytermb + ytermc + ytermd) * rma;
      vz[ia] -= (ztermb + ztermc + ztermd) * rma;
      vx[ib] += xtermb * rmb;
      vy[ib] += ytermb * rmb;
      vz[ib] += ztermb * rmb;
      vx[ic] += xtermc * rmc;
      vy[ic] += ytermc * rmc;
      vz[ic] += ztermc * rmc;
      if (methyl) {
         vx[id] += xtermd * rmd;
         vy[id] += ytermd * rmd;
         vz[id] += ztermd * rmd;
      }
      if CONSTEXPR (DO_V) {
         xtermb *= vterm;
         ytermb *= vterm;
         ztermb *= vterm;
         xtermc *= vterm;
         ytermc *= vterm;
         ztermc *= vterm;
         xtermd *= vterm;
         ytermd *= vterm;
         ztermd *= vterm;
         vxx -= (xb3 * xtermb + xc3 * xtermc + xd3 * xtermd);
         vyx -= (yb3 * xtermb + yc3 * xtermc + yd3 * xtermd);
         vzx -= (zb3 * xtermb + zc3 * xtermc + zd3 * xtermd);
         vyy -= (yb3 * ytermb + yc3 * ytermc + yd3 * ytermd);
         vzz -= (zb3 * ztermb + zc3 * ztermc + zd3 * ztermd);
         vzy -= (zb3 * ytermb + zc3 * ytermc + zd3 * ytermd);
      }
   }


   if CONSTEXPR (DO_V) {
      atomic_add((real)vxx, (real)vyx, (real)vzx, (real)vyy, (real)vzy,
                 (real)vzz, vir_buf, ithread);
   }
}


void rattle2_methyl_cu(time_prec dt, bool do_v)
{
   int n23 = nratch2 + nratch3;
   if (n23 <= 0)
      return;


   if (do_v) {
      auto ker = constrain2_methyl_cu1<true>;
      launch_k2b(g::s0, 64, n23, ker,

                 nratch2, iratch2, nratch3, iratch3,

                 dt, vx, vy, vz, vir_buf,

                 xpos, ypos, zpos, massinv);
   } else {
      auto ker = constrain2_methyl_cu1<false>;
      launch_k2b(g::s0, 64, n23, ker,

                 nratch2, iratch2, nratch3, iratch3,

                 dt, vx, vy, vz, nullptr,

                 xpos, ypos, zpos, massinv);
   }
}
}
