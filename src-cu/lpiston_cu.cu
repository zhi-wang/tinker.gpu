#include "hip/hip_runtime.h"
#include "add.h"
#include "launch.h"
#include "lpiston.h"
#include "mdegv.h"
#include "mdpq.h"
#include "tool/darray.h"
#include <tinker/detail/units.hh>


namespace tinker {
__global__
void lp_mol_virial_cu1(virial_buffer restrict lp_vir_buf,

                       const double* restrict mass,
                       const pos_prec* restrict xpos,
                       const pos_prec* restrict ypos,
                       const pos_prec* restrict zpos,
                       const grad_prec* restrict gx,
                       const grad_prec* restrict gy,
                       const grad_prec* restrict gz,

                       int nmol, const int (*restrict imol)[2],
                       const int* restrict kmol, const double* restrict molmass)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;

   for (int im = ithread; im < nmol; im += stride) {
      double vxx = 0, vyy = 0, vzz = 0, vxy = 0, vxz = 0, vyz = 0;
      double igx, igy, igz;             // atomic gradients
      pos_prec irx, iry, irz;           // atomic positions
      double mgx = 0, mgy = 0, mgz = 0; // molecular gradients
      pos_prec rx = 0, ry = 0, rz = 0;  // molecular positions
      int start = imol[im][0];
      int end = imol[im][1];
      for (int i = start; i < end; ++i) {
         int k = kmol[i];
#if TINKER_DETERMINISTIC_FORCE
         igx = to_flt_cu<double>(gx[k]);
         igy = to_flt_cu<double>(gy[k]);
         igz = to_flt_cu<double>(gz[k]);
#else
         igx = gx[k];
         igy = gy[k];
         igz = gz[k];
#endif
         irx = xpos[k];
         iry = ypos[k];
         irz = zpos[k];
         vxx -= igx * irx;
         vyy -= igy * iry;
         vzz -= igz * irz;
         vxy -= 0.5 * (igx * iry + igy * irx);
         vxz -= 0.5 * (igx * irz + igz * irx);
         vyz -= 0.5 * (igy * irz + igz * iry);

         mgx += igx;
         mgy += igy;
         mgz += igz;
         auto massk = mass[k];
         rx += massk * irx;
         ry += massk * iry;
         rz += massk * irz;
      }
      auto mmassinv = 1 / molmass[im];
      vxx += mgx * rx * mmassinv;
      vyy += mgy * ry * mmassinv;
      vzz += mgz * rz * mmassinv;
      vxy += 0.5 * (mgx * ry + mgy * rx) * mmassinv;
      vxz += 0.5 * (mgx * rz + mgz * rx) * mmassinv;
      vyz += 0.5 * (mgy * rz + mgz * ry) * mmassinv;
      atomic_add(vxx, vxy, vxz, vyy, vyz, vzz, lp_vir_buf, ithread);
   }
}


void lp_mol_virial_cu()
{
   auto bufsize = buffer_size();
   darray::zero(g::q0, bufsize, lp_vir_buf);

   launch_k1b(g::s0, n, lp_mol_virial_cu1,

              lp_vir_buf, mass, xpos, ypos, zpos, gx, gy, gz,

              rattle_dmol.nmol, rattle_dmol.imol, rattle_dmol.kmol,
              rattle_dmol.molmass);

   virial_reduce(lp_vir, lp_vir_buf);
   for (int iv = 0; iv < 9; ++iv)
      lp_vir[iv] += vir[iv];
}
}
