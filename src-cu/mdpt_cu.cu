#include "hip/hip_runtime.h"
#include "launch.h"
#include "md.h"
#include "syntax/cu/reduce.h"
#include "tool/gpu_card.h"
#include <tinker/detail/units.hh>


namespace tinker {
// velocity to ekin[3][3] (actually ekin[6])
template <unsigned int B>
__global__
void velocity_to_ekin_cu(energy_prec* out, const vel_prec* restrict vx,
                         const vel_prec* restrict vy,
                         const vel_prec* restrict vz,
                         const double* restrict mass, int n,
                         energy_prec ekcal_inv)
{
   constexpr int HN = 6;
   __shared__ energy_prec sd[HN][B];
   unsigned int t = threadIdx.x;
   #pragma unroll
   for (int j = 0; j < HN; ++j)
      sd[j][t] = 0;
   for (int i = t + blockIdx.x * B; i < n; i += B * gridDim.x) {
      energy_prec term = 0.5f * mass[i] * ekcal_inv;
      sd[0][t] += term * vx[i] * vx[i]; // exx
      sd[1][t] += term * vy[i] * vy[i]; // eyy
      sd[2][t] += term * vz[i] * vz[i]; // ezz
      sd[3][t] += term * vx[i] * vy[i]; // exy
      sd[4][t] += term * vy[i] * vz[i]; // eyz
      sd[5][t] += term * vz[i] * vx[i]; // ezx
   }
   __syncthreads();


   using Op = OpPlus<energy_prec>;
   Op op;
   static_assert(B <= 512, "");
   // clang-format off
   if (B >= 512) { if (t < 256) { _Pragma("unroll") for (int j = 0; j < HN; ++j) sd[j][t] = op(sd[j][t], sd[j][t + 256]); } __syncthreads(); }
   if (B >= 256) { if (t < 128) { _Pragma("unroll") for (int j = 0; j < HN; ++j) sd[j][t] = op(sd[j][t], sd[j][t + 128]); } __syncthreads(); }
   if (B >= 128) { if (t < 64 ) { _Pragma("unroll") for (int j = 0; j < HN; ++j) sd[j][t] = op(sd[j][t], sd[j][t + 64 ]); } __syncthreads(); }
   if (t < 32  ) warp_reduce2<energy_prec, HN, B, Op>(sd, t, op);
   // clang-format on
   if (t == 0)
      #pragma unroll
      for (int j = 0; j < HN; ++j)
         out[blockIdx.x * HN + j] = sd[j][0];
}


void kinetic_energy_cu(energy_prec& eksum_out, energy_prec (&ekin_out)[3][3],
                       int n, const double* mass, const vel_prec* vx,
                       const vel_prec* vy, const vel_prec* vz)
{
   hipStream_t st = g::s0;
   constexpr int HN = 6;
   energy_prec* dptr = reinterpret_cast<energy_prec*>(dptr_buf);
   energy_prec(*dptr6)[HN] = reinterpret_cast<energy_prec(*)[HN]>(dptr_buf);
   energy_prec* hptr = reinterpret_cast<energy_prec*>(pinned_buf);
   int grid_siz1 = get_grid_size(BLOCK_DIM);
   grid_siz1 = grid_siz1 / HN;
   int grid_siz2 = (n + BLOCK_DIM - 1) / BLOCK_DIM;
   int grid_size = std::min(grid_siz1, grid_siz2);
   const energy_prec ekcal_inv = 1.0 / units::ekcal;
   velocity_to_ekin_cu<BLOCK_DIM>
      <<<grid_size, BLOCK_DIM, 0, st>>>(dptr, vx, vy, vz, mass, n, ekcal_inv);
   reduce2<energy_prec, BLOCK_DIM, HN, HN, OpPlus<energy_prec>>
      <<<1, BLOCK_DIM, 0, st>>>(dptr6, dptr6, grid_size);
   check_rt(hipMemcpyAsync(hptr, dptr, HN * sizeof(energy_prec),
                            hipMemcpyDeviceToHost, st));
   check_rt(hipStreamSynchronize(st));
   energy_prec exx = hptr[0];
   energy_prec eyy = hptr[1];
   energy_prec ezz = hptr[2];
   energy_prec exy = hptr[3];
   energy_prec eyz = hptr[4];
   energy_prec ezx = hptr[5];


   ekin_out[0][0] = exx;
   ekin_out[0][1] = exy;
   ekin_out[0][2] = ezx;
   ekin_out[1][0] = exy;
   ekin_out[1][1] = eyy;
   ekin_out[1][2] = eyz;
   ekin_out[2][0] = ezx;
   ekin_out[2][1] = eyz;
   ekin_out[2][2] = ezz;
   eksum_out = exx + eyy + ezz;
}
}
